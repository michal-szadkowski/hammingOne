#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <cstring>
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <stdint.h>
#include <chrono>
#include <iomanip>
#include "fileIO.h"

void usage();

void getArguments(int argc, char *argv[], std::string &outputFile, char *&vectors, unsigned int &vecCount, unsigned int &vecLen);

//#define VERBOSEPAIRS
__global__ void hammingOne2(char *vectors, int vecLenB, int vecCount, unsigned int *pairCount)
{

	int pairs = 0;
	int vecI = blockIdx.x * blockDim.x + threadIdx.x;
	if (vecI >= vecCount)
		return;

	char *v1 = new char[vecLenB];
	for (int i = 0; i < vecLenB; i++)
	{

		v1[i] = vectors[i * vecCount + vecI];
	}

	if (vecI < vecCount)
	{
		int v2 = vecI + 1;

		for (int c = 0; 2 * c <= vecCount; c++)
		{
			if (v2 >= vecCount)
			{
				v2 -= vecCount;
				c++;
				c++;
				continue;
			}

			int d = 0;
			for (int i = 0; i < vecLenB; i++)
			{
				char c = v1[i] ^ vectors[i * vecCount + v2];

				d += __popc(c & 255);
				if (d > 1)
					break;
			}
			if (d == 1)
			{
				pairs++;
			}

			v2++;
		}
	}
	delete v1;
	atomicAdd(pairCount, pairs);
	return;
}
void timeElapsedCout(float start, float event, std::string name)
{
	std::cout << std::left << std::setw(28) << name + " ended,"
			  << "time elapsed:" << event << " ms, \tsince start: " << start << "ms\n";
}

void eventStartMeasure(hipEvent_t start, std::string name)
{
	std::cout << name << " started\n";
	hipEventRecord(start, 0);
}
void eventStopMeasure(hipEvent_t stop, hipEvent_t start, float &startms, std::string name)
{
	float evms;
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();

	hipEventElapsedTime(&evms, start, stop);
	startms += evms;
	timeElapsedCout(startms, evms, name);
}

int main(int argc, char *argv[])
{
	std::string outputFile = "";
	unsigned int vecCount, vecLen;

	char *vectors;
	char *d_vectors;
	unsigned int *d_pairCount;

	getArguments(argc, argv, outputFile, vectors, vecCount, vecLen);

	unsigned int vecLenBytes = std::ceil(double(vecLen) / 8.0);

	char *vectorsCoales = new char[vecLenBytes * vecCount];
	for (int i = 0; i < vecCount; i++)
	{
		for (int j = 0; j < vecLenBytes; j++)
		{
			vectorsCoales[j * vecCount + i] = vectors[i * vecLenBytes + j];
		}
	}

	dim3 blockSize2(1024);
	dim3 blockCount2(ceil((double)vecCount / blockSize2.x));

	hipEvent_t p1, p2;
	float startms = 0;
	std::cout << "tyutaj";
	hipEventCreate(&p1);
	hipEventCreate(&p2);

	eventStartMeasure(p1, "Memory allocation");
	hipMalloc(&d_vectors, vecCount * vecLenBytes);
	hipMalloc(&d_pairCount, sizeof(int));
	eventStopMeasure(p2, p1, startms, "Memory allocation");

	eventStartMeasure(p1, "Memory copy");
	hipMemcpy(d_vectors, vectorsCoales, vecCount * vecLenBytes, hipMemcpyHostToDevice);
	eventStopMeasure(p2, p1, startms, "Memory copy");

	eventStartMeasure(p1, "Kernel");
	hammingOne2<<<blockCount2, blockSize2>>>(d_vectors, vecLenBytes, vecCount, d_pairCount);
	eventStopMeasure(p2, p1, startms, "Kernel");

	eventStartMeasure(p1, "Free memory");
	hipFree(d_vectors);
	eventStopMeasure(p2, p1, startms, "Free memory");

	unsigned int pairCount;
	hipMemcpy(&pairCount, d_pairCount, 4, hipMemcpyDeviceToHost);
	hipFree(d_pairCount);
	std::cout << "Pair count: " << pairCount << "\n";
	if (outputFile != "")
	{
		writeDataFile(outputFile, startms, vecCount, vecLen);
	}

	delete[] vectors;
	delete[] vectorsCoales;

	return 0;
}

void usage()
{
	std::cout << "usage: hamGPU inputfile [outputfile]\n";
	exit(0);
}

void getArguments(int argc, char *argv[], std::string &outputFile, char *&vectors, unsigned int &vecCount, unsigned int &vecLen)
{
	if (argc < 2 || argc > 3)
		usage();

	vectors = readFile(argv[1], vecCount, vecLen);
	if (vectors == NULL)
		usage();
	if (argc >= 3)
		outputFile = argv[2];
}